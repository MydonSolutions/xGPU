#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <complex>
#include <limits.h>
#include <omp.h>

/*
  Data ordering for input vectors is (running from slowest to fastest)
  [time][channel][station][polarization][complexity]

  Output matrix has ordering
  [channel][station][station][polarization][polarization][complexity]
*/

#define USE_GPU

// set the data type accordingly
typedef std::complex<char> ComplexInput;
#define COMPLEX_INPUT char2 
#define SCALE 1 // no need to rescale result 

#define TRIANGULAR_ORDER 1000
#define REAL_IMAG_TRIANGULAR_ORDER 2000
#define REGISTER_TILE_TRIANGULAR_ORDER 3000
#define MATRIX_ORDER REGISTER_TILE_TRIANGULAR_ORDER

// size = freq * time * station * pol *sizeof(ComplexInput)
#define GBYTE (1024llu*1024llu*1024llu)

#define NPOL 2
#define NSTATION 256ll
#define SIGNAL_SIZE GBYTE
#define SAMPLES SIGNAL_SIZE / (NSTATION*NPOL*sizeof(ComplexInput))
#define NFREQUENCY 10ll
#define NTIME 1000ll //SAMPLES / NFREQUENCY
#define NBASELINE ((NSTATION+1)*(NSTATION/2))
#define NDIM 2

//#define PIPE_LENGTH 1
//#define NTIME_PIPE NTIME / PIPE_LENGTH

#define NTIME_PIPE 100
#define PIPE_LENGTH NTIME / NTIME_PIPE

// how many pulsars are we binning for (Not implemented yet)
#define NPULSAR 0

// whether we are writing the matrix back to device memory (used for benchmarking)
int writeMatrix = 1;
// this must be enabled for this option to work though, slightly hurts performance
//#define WRITE_OPTION 

typedef std::complex<int> Complex;

Complex convert(const ComplexInput &b) {
  return Complex(real(b), imag(b));
}

// the OpenMP Xengine
#include "omp_xengine.cc"

// the GPU Xengine
#include "cuda_xengine.cu"

#include "cpu_util.cc"

int main(int argc, char** argv) {

  unsigned int seed = 1;
  int verbose = 0;

  if(argc>1) {
    seed = strtoul(argv[1], NULL, 0);
  }
  if(argc>2) {
    verbose = strtoul(argv[2], NULL, 0);
  }

  srand(seed);

  printf("Correlating %llu stations with %llu signals, with %llu channels and integration length %llu\n",
	 NSTATION, SAMPLES, NFREQUENCY, NTIME);

  unsigned long long vecLength = NFREQUENCY * NTIME * NSTATION * NPOL;


  // perform host memory allocation
  int packedMatLength = NFREQUENCY * ((NSTATION+1)*(NSTATION/2)*NPOL*NPOL);

  // allocate the GPU X-engine memory
  ComplexInput *array_h = 0; // this is pinned memory
  Complex *cuda_matrix_h = 0;
  xInit(&array_h, &cuda_matrix_h, NSTATION);

  // create an array of complex noise
  random_complex(array_h, vecLength);

  Complex *omp_matrix_h = (Complex *) malloc(packedMatLength*sizeof(Complex));
  printf("Calling CPU X-Engine\n");
#if (CUBE_MODE == CUBE_DEFAULT)
  ompXengine(omp_matrix_h, array_h);
#endif

  printf("Calling GPU X-Engine\n");
  cudaXengine(cuda_matrix_h, array_h);

#if (CUBE_MODE == CUBE_DEFAULT)
  
  reorderMatrix(cuda_matrix_h);
  checkResult(cuda_matrix_h, omp_matrix_h, verbose, array_h);

  int fullMatLength = NFREQUENCY * NSTATION*NSTATION*NPOL*NPOL;
  Complex *full_matrix_h = (Complex *) malloc(fullMatLength*sizeof(Complex));

  // convert from packed triangular to full matrix
  extractMatrix(full_matrix_h, cuda_matrix_h);

  free(full_matrix_h);
#endif

  //free host memory
  free(omp_matrix_h);

  // free gpu memory
  xFree(array_h, cuda_matrix_h);

  return 0;
}
